#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"
#include ""
#include <math.h>

__global__ void kernel_A(float *g_data)
{
  int idx = blockDim.x*blockIdx.x + threadIdx.x;
  //���˷������滻Ϊλ���������Ч��
  idx <<= 1;
  //����һ���̴߳������ڵ�������ż���ݣ����������ж�����ִ�У�Ҳ����ʹÿ���߳��������ܶ������
  float value0 = g_data[idx];
  float value1 = g_data[idx + 1];

  value0 += sqrtf(cosf(value0) + 1.f);
  value1 += sqrtf(logf(value1) + 1.f);

  g_data[idx] = value0;
  g_data[idx + 1] = value1;
}

float timing_experiment( void (*kernel)( float*), float *d_data, int dimx, int dimy, int nreps, int blockx, int blocky )
{
	float elapsed_time_ms=0.0f;
	hipEvent_t start, stop;
	hipEventCreate( &start );
	hipEventCreate( &stop  );

  //����block��grid��С�Ͳ��֣��������ݵĶ�ȡ����;ֲ���ԭ�����ң�����d_data��һά���ݣ�����һά��block��gridЧ������
  int block = 512;
  int grid = dimx*dimy / (block<<1);

	hipEventRecord( start, 0 );
	for(int i=0; i<nreps; i++)	// do not change this loop, it's not part of the algorithm - it's just to average time over several kernel launches
		kernel<<<grid,block>>>( d_data);  //�����߳����̶�����֪������£�ȥ��dimy,dimx�����ٲ���Ҫ��launch��������
	hipEventRecord( stop, 0 );
	hipDeviceSynchronize();
  //hipEventSynchronize(stop);
	hipEventElapsedTime( &elapsed_time_ms, start, stop );
	elapsed_time_ms /= nreps;

	hipEventDestroy( start );
	hipEventDestroy( stop );

	return elapsed_time_ms;
}

int main()
{
	int dimx = 2*1024;
	int dimy = 2*1024;

	int nreps = 10;

	int nbytes = dimx*dimy*sizeof(float);

	float *d_data=0, *h_data=0;
	hipMalloc( (void**)&d_data, nbytes );
	if( 0 == d_data )
	{
		printf("couldn't allocate GPU memory\n");
		return -1;
	}
	printf("allocated %.2f MB on GPU\n", nbytes/(1024.f*1024.f) );
	h_data = (float*)malloc( nbytes );
	if( 0 == h_data )
	{
		printf("couldn't allocate CPU memory\n");
		return -2;
	}
	printf("allocated %.2f MB on CPU\n", nbytes/(1024.f*1024.f) );
	for(int i=0; i<dimx*dimy; i++)
		h_data[i] = 10.f + rand() % 256;
	hipMemcpy( d_data, h_data, nbytes, hipMemcpyHostToDevice );

	float elapsed_time_ms=0.0f;

	elapsed_time_ms = timing_experiment( kernel_A, d_data, dimx,dimy, nreps, 1, 512 );
	printf("A:  %8.2f ms\n", elapsed_time_ms );

	printf("CUDA: %s\n", hipGetErrorString( hipGetLastError() ) );

	if( d_data )
		hipFree( d_data );
	if( h_data )
		free( h_data );

	hipDeviceReset();

	return 0;
}