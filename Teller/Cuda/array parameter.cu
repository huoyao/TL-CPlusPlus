#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include <stdio.h>

__device__ void ar(int k[])
{
  for(int i = 0;i < 10;++i)
  {
    k[i] = i;
  }
}

__global__ void array(int n)
{
  int idx = blockDim.x*blockIdx.x + threadIdx.x;
  if(idx >= n) return;
  int k[10];
  ar(k);
  int i = k[0];
  printf("%d\n",k[0]);
}

int main()
{
  array<<<1,10>>>(10);
  return;
}