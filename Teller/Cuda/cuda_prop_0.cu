#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"
#include ""
#include <math.h>

__global__ void kernel_A_(float *g_data)
{
  int idx = blockDim.x*blockIdx.x + threadIdx.x;
  //���˷������滻Ϊλ���������Ч��
  idx <<= 1;
  //����һ���̴߳������ڵ�������ż���ݣ����������ж�����ִ�У�Ҳ����ʹÿ���߳��������ܶ������
  float value0 = g_data[idx];
  float value1 = g_data[idx + 1];

  value0 += sqrtf(cosf(value0) + 1.f);
  value1 += sqrtf(logf(value1) + 1.f);

  g_data[idx] = value0;
  g_data[idx + 1] = value1;
}

float timing_experiment_(void(*kernel)(float*), float *d_data, int dimx, int dimy, int nreps, int blockx, int blocky)
{
  float elapsed_time_ms = 0.0f;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  //����block��grid��С���������ݵĶ�ȡ����;ֲ���ԭ�����ң�����d_data��һά���ݣ�����һά��block��gridЧ������
  int block = 512;
  int grid = dimx*dimy / (block << 1);

  hipEventRecord(start, 0);
  for(int i = 0; i<nreps; i++)	// do not change this loop, it's not part of the algorithm - it's just to average time over several kernel launches
    kernel << <grid, block >> >(d_data);  //�����߳�����֪������£�ȥ��dimy,dimx�����ٲ���Ҫ�Ĳ�������
  hipEventRecord(stop, 0);
  hipDeviceSynchronize();
  //hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsed_time_ms, start, stop);
  elapsed_time_ms /= nreps;

  hipEventDestroy(start);
  hipEventDestroy(stop);

  return elapsed_time_ms;
}

__global__ void kernel_A(float *g_data, int dimx, int dimy)
{
  int ix = blockIdx.x;
  int iy = blockIdx.y*blockDim.y + threadIdx.y;
  int idx = iy*dimx + ix;

  float value = g_data[idx];

  if(ix % 2)
  {
    value += sqrtf(logf(value) + 1.f);
  } else
  {
    value += sqrtf(cosf(value) + 1.f);
  }

  g_data[idx] = value;
}

float timing_experiment(void(*kernel)(float*, int, int), float *d_data, int dimx, int dimy, int nreps, int blockx, int blocky)
{
  float elapsed_time_ms = 0.0f;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  //blockx = 512;
  dim3 block(blockx, blocky);
  dim3 grid(dimx / block.x, dimy / block.y);
  //int block = 128;
  //int grid = dimx*dimy / block;

  hipEventRecord(start, 0);
  for(int i = 0; i < nreps; i++)	// do not change this loop, it's not part of the algorithm - it's just to average time over several kernel launches
    kernel << <grid, block >> >(d_data, dimx, dimy);
  hipEventRecord(stop, 0);
  hipDeviceSynchronize();
  hipEventElapsedTime(&elapsed_time_ms, start, stop);
  elapsed_time_ms /= nreps;

  hipEventDestroy(start);
  hipEventDestroy(stop);

  return elapsed_time_ms;
}

__global__ void kerl(float *d,float *d_)
{
  printf("");
}

int main()
{
  int dimx = 2 * 1024;
  int dimy = 2 * 1024;

  int nreps = 10;

  int nbytes = dimx*dimy*sizeof(float);

  float *d_data = 0, *h_data = 0,*d_data_;
  hipMalloc((void**)&d_data, nbytes);
  hipMalloc((void**)&d_data_, nbytes);
  if(0 == d_data)
  {
    printf("couldn't allocate GPU memory\n");
    return -1;
  }
  printf("allocated %.2f MB on GPU\n", nbytes / (1024.f*1024.f));
  h_data = (float*)malloc(nbytes);
  if(0 == h_data)
  {
    printf("couldn't allocate CPU memory\n");
    return -2;
  }
  printf("allocated %.2f MB on CPU\n", nbytes / (1024.f*1024.f));
  for(int i = 0; i < dimx*dimy; i++)
    h_data[i] = 10.f + rand() % 256;
  hipMemcpy(d_data, h_data, nbytes, hipMemcpyHostToDevice);
  hipMemcpy(d_data_, h_data, nbytes, hipMemcpyHostToDevice);

  float elapsed_time_ms = 0.0f;

  elapsed_time_ms = timing_experiment(kernel_A, d_data, dimx, dimy, nreps, 1, 512);
  printf("A:  %8.2f ms\n", elapsed_time_ms);

  float elapsed_time_ms_ = 0.0f;

  elapsed_time_ms_ = timing_experiment_(kernel_A_, d_data_, dimx, dimy, nreps, 1, 512);
  printf("A:  %8.2f ms\n", elapsed_time_ms_);

  printf("CUDA: %s\n", hipGetErrorString(hipGetLastError()));

  float *h_data0, *h_data1;
  h_data0 = (float*)malloc(nbytes);
  h_data1 = (float*)malloc(nbytes);
  hipMemcpy(h_data0, d_data, nbytes, hipMemcpyDeviceToHost);
  hipMemcpy(h_data1, d_data_, nbytes, hipMemcpyDeviceToHost);
  kerl << <1, 1 >> >(d_data, d_data_);
  for(size_t i = 0; i < dimx*dimy; i++)
  {
    if(abs(h_data0[i] - h_data1[i]>1e-6))
      printf("%f %f\n", h_data0[i],h_data1[i]);
  }
  if(d_data)
    hipFree(d_data);
  if(h_data)
    free(h_data);

  hipDeviceReset();

  return 0;
}