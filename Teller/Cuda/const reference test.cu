#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <thrust/device_vector.h>

__device__ void test_ref(const int& t)
{
  printf("%d\n", t);
}

__device__ void test_ref_(int &t)
{
  ++t;
}

__global__ void test_const()
{
  const int i = 3;
  test_ref(i);
  int t = 4;
  test_ref_(t);
  printf("%d %d\n",i, t);
}

int main()
{
  test_const << <1, 1 >> >();
  return 0;
}