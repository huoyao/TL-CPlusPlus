#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"
#include ""
#include <math.h>

__global__ void kernel_A(float *g_data, int dimx, int dimy)
{
  int ix = blockIdx.x;
  int iy = blockIdx.y*blockDim.y + threadIdx.y;
  int idx = iy*dimx + ix;

  float value = g_data[idx];

  if(ix % 2)
  {
    value += sqrtf(logf(value) + 1.f);
  } else
  {
    value += sqrtf(cosf(value) + 1.f);
  }

  g_data[idx] = value;
}

float timing_experiment(void(*kernel)(float*, int, int), float *d_data, int dimx, int dimy, int nreps, int blockx, int blocky)
{
  float elapsed_time_ms = 0.0f;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  //blockx = 512;
  dim3 block(blockx, blocky);
  dim3 grid(dimx / block.x, dimy / block.y);
  //int block = 128;
  //int grid = dimx*dimy / block;

  hipEventRecord(start, 0);
  for(int i = 0; i < nreps; i++)	// do not change this loop, it's not part of the algorithm - it's just to average time over several kernel launches
    kernel << <grid, block >> >(d_data, dimx, dimy);
  hipEventRecord(stop, 0);
  hipDeviceSynchronize();
  hipEventElapsedTime(&elapsed_time_ms, start, stop);
  elapsed_time_ms /= nreps;

  hipEventDestroy(start);
  hipEventDestroy(stop);

  return elapsed_time_ms;
}

int main1()
{
  int dimx = 2 * 1024;
  int dimy = 2 * 1024;

  int nreps = 10;

  int nbytes = dimx*dimy*sizeof(float);

  float *d_data = 0, *h_data = 0;
  hipMalloc((void**)&d_data, nbytes);
  if(0 == d_data)
  {
    printf("couldn't allocate GPU memory\n");
    return -1;
  }
  printf("allocated %.2f MB on GPU\n", nbytes / (1024.f*1024.f));
  h_data = (float*)malloc(nbytes);
  if(0 == h_data)
  {
    printf("couldn't allocate CPU memory\n");
    return -2;
  }
  printf("allocated %.2f MB on CPU\n", nbytes / (1024.f*1024.f));
  for(int i = 0; i < dimx*dimy; i++)
    h_data[i] = 10.f + rand() % 256;
  hipMemcpy(d_data, h_data, nbytes, hipMemcpyHostToDevice);

  float elapsed_time_ms = 0.0f;

  elapsed_time_ms = timing_experiment(kernel_A, d_data, dimx, dimy, nreps, 1, 512);
  printf("A:  %8.2f ms\n", elapsed_time_ms);

  printf("CUDA: %s\n", hipGetErrorString(hipGetLastError()));

  if(d_data)
    hipFree(d_data);
  if(h_data)
    free(h_data);

  hipDeviceReset();

  return 0;
}