#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include "hip/hip_vector_types.h"
#include <iostream>
#include ""
#include <iostream>
#include "stdio.h"
using namespace std;

class Base
{
public:
  int *dt;
};

class Ds
{
public:
  __device__ __host__ void getData(int []);
  __device__ __host__ void printData();
  Base bs;
  int *data;
};

__device__ __host__ void Ds::getData(int bb[])
{
  bb[0]=1;
}

__device__ __host__ void printData(int *data)
{
  printf("%d\n",data[0]);
}

__global__ void kernel_print(Ds *ds)
{
  int bk[3];
  ds->getData(bk);
  //ds->data[0] = 1;
  printData(ds->data);
  printf("%d\n",bk[0]);
}

__global__ void kernel_test(int *ds)
{
  //int *bk = ds->getData();
  //ds->getData();
  //ds->bs.dt = bk;
}

int main()
{
  Ds *ds = new Ds();
  int *dint = NULL;
  checkCudaErrors(hipMalloc(&ds->data, sizeof(int)* 3));
  checkCudaErrors(hipMemset(ds->data, 0, sizeof(int)* 3));
  kernel_print<<<1,1>>>(ds);
  kernel_test << <1, 1 >> >(dint);
  return 0;
}