#include "hip/hip_runtime.h"
#include <stdio.h>

__device__ void ar(int k[])
{
  for(int i = 0;i < 10;++i)
  {
    k[i] = i;
  }
}

__global__ void array()
{
  int k[10];
  ar(k);
  int i = k[0];
  printf("%d\n",k[0]);
}

int main()
{
  array<<<1,1>>>();
  return;
}