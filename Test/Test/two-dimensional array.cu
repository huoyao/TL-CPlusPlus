#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_vector_types.h"
#include <iostream>
#include ""
#include <hip/hip_runtime_api.h>
#include <iostream>
#include "stdio.h"
#include "atomic.cuh"
using namespace std;

struct mat
{
  double3 col0;
  double3 col1;
  double3 col2;
  //__device__ ~mat(){};
};

__device__ void adda(int **a)  
{
  a[0][0] = 1;
}

__global__ void kernel_test(int **a)
{
  adda(a);
  adda(a);
  double3 d3;
  atomicAdd(&d3.x,1.);
  printf("ok\n");
  printf("%d\n",a[0][0]);
}

int main()
{
  const int ROWS = 3,COLS = 3;
  int **da = NULL,**ha=NULL;
  int *dc = NULL;

  hipMalloc((void**)(&da), ROWS*sizeof(int*));
  hipMalloc((void**)(&dc), ROWS*COLS*sizeof(int));
  ha = (int**)malloc(ROWS*sizeof(int*));
  (int*)malloc(ROWS*COLS*sizeof(int));
  for(int r = 0; r < ROWS; r++)
  {
    ha[r] = dc + r*COLS;
  }
  hipMemcpy(da,ha, ROWS*sizeof(int*), hipMemcpyHostToDevice);
  kernel_test <<<1, 1>>>(da);
  return 0;
}